#include "helix/cuda/cuda_flat.hpp"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

namespace helix {
namespace cuda_utils {

bool isCudaAvailable() {
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess) {
        return false;
    }
    return deviceCount > 0;
}

int getDeviceCount() {
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess) {
        return 0;
    }
    return deviceCount;
}

void setDevice(int deviceId) {
    hipError_t error = hipSetDevice(deviceId);
    if (error != hipSuccess) {
        throw HelixException("Failed to set CUDA device: " + std::string(hipGetErrorString(error)));
    }
}

size_t getFreeMemory() {
    size_t free, total;
    hipError_t error = hipMemGetInfo(&free, &total);
    if (error != hipSuccess) {
        return 0;
    }
    return free;
}

size_t getTotalMemory() {
    size_t free, total;
    hipError_t error = hipMemGetInfo(&free, &total);
    if (error != hipSuccess) {
        return 0;
    }
    return total;
}

void synchronize() {
    hipError_t error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        throw HelixException("CUDA synchronization failed: " + std::string(hipGetErrorString(error)));
    }
}

void* allocate(size_t size) {
    void* ptr;
    hipError_t error = hipMalloc(&ptr, size);
    if (error != hipSuccess) {
        throw HelixException("CUDA memory allocation failed: " + std::string(hipGetErrorString(error)));
    }
    return ptr;
}

void free(void* ptr) {
    if (ptr) {
        hipError_t error = hipFree(ptr);
        if (error != hipSuccess) {
            throw HelixException("CUDA memory free failed: " + std::string(hipGetErrorString(error)));
        }
    }
}

void memcpyHtoD(void* dst, const void* src, size_t size) {
    hipError_t error = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        throw HelixException("CUDA memcpy HtoD failed: " + std::string(hipGetErrorString(error)));
    }
}

void memcpyDtoH(void* dst, const void* src, size_t size) {
    hipError_t error = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        throw HelixException("CUDA memcpy DtoH failed: " + std::string(hipGetErrorString(error)));
    }
}

void memcpyAsync(void* dst, const void* src, size_t size, hipStream_t stream) {
    hipError_t error = hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, stream);
    if (error != hipSuccess) {
        throw HelixException("CUDA memcpy async failed: " + std::string(hipGetErrorString(error)));
    }
}

// Additional utility functions
void checkCudaError(const char* file, int line) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::string msg = "CUDA error at " + std::string(file) + ":" + std::to_string(line) + 
                         " - " + std::string(hipGetErrorString(error));
        throw HelixException(msg);
    }
}

void printDeviceInfo() {
    int deviceCount = getDeviceCount();
    std::cout << "CUDA Device Count: " << deviceCount << std::endl;
    
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        
        std::cout << "Device " << i << ": " << prop.name << std::endl;
        std::cout << "  Compute Capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "  Total Memory: " << prop.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Multiprocessors: " << prop.multiProcessorCount << std::endl;
        std::cout << "  Max Threads per Block: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "  Max Threads per Multiprocessor: " << prop.maxThreadsPerMultiProcessor << std::endl;
    }
}

bool isDeviceCompatible(int deviceId) {
    if (deviceId < 0 || deviceId >= getDeviceCount()) {
        return false;
    }
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceId);
    
    // Check compute capability (require 7.5+ for optimal performance)
    return prop.major > 7 || (prop.major == 7 && prop.minor >= 5);
}

void optimizeDevice(int deviceId) {
    if (deviceId < 0 || deviceId >= getDeviceCount()) {
        throw HelixException("Invalid device ID for optimization");
    }
    
    hipSetDevice(deviceId);
    
    // Set device flags for optimal performance
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
}

}
}
