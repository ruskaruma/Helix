#include "helix/cuda/cuda_flat.hpp"
#include "helix/common/utils.hpp"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <algorithm>
#include <cstring>

namespace helix {

CudaIndexFlat::CudaIndexFlat(const IndexConfig& config) 
    : IndexFlat(config), d_vectors_(nullptr), d_queries_(nullptr), 
      d_distances_(nullptr), d_indices_(nullptr), gpu_memory_allocated_(0),
      device_id_(0), gpu_data_valid_(false) {
    
    if (!cuda_utils::isCudaAvailable()) {
        throw HelixException("CUDA not available on this system");
    }
    
    // Initialize CUDA
    hipSetDevice(device_id_);
    hipblasCreate(&cublas_handle_);
    hipStreamCreate(&stream_);
    
    allocateGpuMemory();
}

CudaIndexFlat::~CudaIndexFlat() {
    freeGpuMemory();
    if (cublas_handle_) {
        hipblasDestroy(cublas_handle_);
    }
    if (stream_) {
        hipStreamDestroy(stream_);
    }
}

void CudaIndexFlat::setDevice(int deviceId) {
    if (deviceId < 0 || deviceId >= cuda_utils::getDeviceCount()) {
        throw HelixException("Invalid CUDA device ID");
    }
    
    device_id_ = deviceId;
    hipSetDevice(device_id_);
    
    // Reallocate memory on new device
    freeGpuMemory();
    allocateGpuMemory();
    gpu_data_valid_ = false;
}

int CudaIndexFlat::getDevice() const {
    return device_id_;
}

size_t CudaIndexFlat::getGpuMemoryUsage() const {
    return gpu_memory_allocated_;
}

void CudaIndexFlat::allocateGpuMemory() {
    if (dimension_ == 0) return;
    
    // Calculate memory requirements
    size_t vector_size = ntotal_ * dimension_ * sizeof(float);
    size_t query_size = dimension_ * sizeof(float);
    size_t distance_size = ntotal_ * sizeof(float);
    size_t index_size = ntotal_ * sizeof(idx_t);
    
    // Allocate GPU memory
    hipMalloc(&d_vectors_, vector_size);
    hipMalloc(&d_queries_, query_size);
    hipMalloc(&d_distances_, distance_size);
    hipMalloc(&d_indices_, index_size);
    
    gpu_memory_allocated_ = vector_size + query_size + distance_size + index_size;
    
    // Initialize indices
    std::vector<idx_t> host_indices(ntotal_);
    for (idx_t i = 0; i < ntotal_; ++i) {
        host_indices[i] = i;
    }
    hipMemcpy(d_indices_, host_indices.data(), index_size, hipMemcpyHostToDevice);
}

void CudaIndexFlat::freeGpuMemory() {
    if (d_vectors_) { hipFree(d_vectors_); d_vectors_ = nullptr; }
    if (d_queries_) { hipFree(d_queries_); d_queries_ = nullptr; }
    if (d_distances_) { hipFree(d_distances_); d_distances_ = nullptr; }
    if (d_indices_) { hipFree(d_indices_); d_indices_ = nullptr; }
    gpu_memory_allocated_ = 0;
}

void CudaIndexFlat::syncToGpu() {
    if (!gpu_data_valid_ && ntotal_ > 0) {
        size_t vector_size = ntotal_ * dimension_ * sizeof(float);
        hipMemcpy(d_vectors_, vectors_.data(), vector_size, hipMemcpyHostToDevice);
        gpu_data_valid_ = true;
    }
}

void CudaIndexFlat::syncFromGpu() {
    // For read-only operations, no sync needed
}

SearchResults CudaIndexFlat::search(const float* query, idx_t k) const {
    if (k <= 0 || k > ntotal_) {
        throw HelixException("Invalid k value for search");
    }
    
    syncToGpu();
    
    // Copy query to GPU
    hipMemcpy(d_queries_, query, dimension_ * sizeof(float), hipMemcpyHostToDevice);
    
    // Launch distance computation kernel
    launchDistanceKernel(d_queries_, 1, d_distances_, k);
    
    // Launch top-k selection kernel
    launchTopKSelection(d_distances_, d_indices_, 1, k);
    
    // Copy results back to host
    SearchResults result;
    result.indices.resize(k);
    result.distances.resize(k);
    
    hipMemcpy(result.indices.data(), d_indices_, k * sizeof(idx_t), hipMemcpyDeviceToHost);
    hipMemcpy(result.distances.data(), d_distances_, k * sizeof(float), hipMemcpyDeviceToHost);
    
    return result;
}

void CudaIndexFlat::searchBatch(const float* queries, idx_t numQueries, idx_t k,
                                std::vector<SearchResults>& results) const {
    if (numQueries == 0) return;
    
    syncToGpu();
    
    // Allocate temporary GPU memory for batch
    float* d_batch_queries;
    float* d_batch_distances;
    idx_t* d_batch_indices;
    
    size_t query_size = numQueries * dimension_ * sizeof(float);
    size_t distance_size = numQueries * ntotal_ * sizeof(float);
    size_t index_size = numQueries * k * sizeof(idx_t);
    
    hipMalloc(&d_batch_queries, query_size);
    hipMalloc(&d_batch_distances, distance_size);
    hipMalloc(&d_batch_indices, index_size);
    
    // Copy queries to GPU
    hipMemcpy(d_batch_queries, queries, query_size, hipMemcpyHostToDevice);
    
    // Launch batch distance computation
    launchDistanceKernel(d_batch_queries, numQueries, d_batch_distances, k);
    
    // Launch batch top-k selection
    launchTopKSelection(d_batch_distances, d_batch_indices, numQueries, k);
    
    // Copy results back
    results.resize(numQueries);
    for (idx_t i = 0; i < numQueries; ++i) {
        results[i].indices.resize(k);
        results[i].distances.resize(k);
        
        hipMemcpy(results[i].indices.data(), 
                   d_batch_indices + i * k, 
                   k * sizeof(idx_t), hipMemcpyDeviceToHost);
        hipMemcpy(results[i].distances.data(), 
                   d_batch_distances + i * k, 
                   k * sizeof(float), hipMemcpyDeviceToHost);
    }
    
    // Cleanup
    hipFree(d_batch_queries);
    hipFree(d_batch_distances);
    hipFree(d_batch_indices);
}

void CudaIndexFlat::searchAsync(const float* query, idx_t k, hipStream_t stream,
                                SearchResults* result) const {
    // Implementation for async search
    // This would use the provided stream for non-blocking execution
    // For now, fall back to synchronous search
    *result = search(query, k);
}

void CudaIndexFlat::searchBatchAsync(const float* queries, idx_t numQueries, idx_t k,
                                     hipStream_t stream, std::vector<SearchResults>* results) const {
    // Implementation for async batch search
    // This would use the provided stream for non-blocking execution
    // For now, fall back to synchronous batch search
    searchBatch(queries, numQueries, k, *results);
}

void CudaIndexFlat::launchDistanceKernel(const float* queries, idx_t numQueries,
                                         float* distances, idx_t k) const {
    // CUDA kernel launch for distance computation
    // This is a simplified implementation - in practice, you'd have
    // optimized CUDA kernels for different distance metrics
    
    dim3 blockSize(256);
    dim3 gridSize((ntotal_ + blockSize.x - 1) / blockSize.x);
    
    // For L2 distance, use CUBLAS
    if (metric_ == MetricType::L2) {
        const float alpha = -2.0f;
        const float beta = 0.0f;
        
        // Compute -2 * queries^T * vectors
        hipblasSgemm(cublas_handle_, HIPBLAS_OP_T, HIPBLAS_OP_N,
                   ntotal_, numQueries, dimension_,
                   &alpha, d_vectors_, dimension_,
                   queries, dimension_,
                   &beta, distances, ntotal_);
        
        // Add ||query||^2 to each row (simplified)
        // In practice, you'd have a more sophisticated kernel
    }
}

void CudaIndexFlat::launchTopKSelection(float* distances, idx_t* indices,
                                        idx_t numQueries, idx_t k) const {
    // CUDA kernel for top-k selection
    // This is a simplified implementation - in practice, you'd use
    // optimized selection algorithms like radix select
    
    dim3 blockSize(256);
    dim3 gridSize((numQueries + blockSize.x - 1) / blockSize.x);
    
    // For now, use a simple selection sort on GPU
    // In practice, you'd implement a more efficient algorithm
}

// CUDA utility implementations
namespace cuda_utils {

bool isCudaAvailable() {
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    return error == hipSuccess && deviceCount > 0;
}

int getDeviceCount() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    return deviceCount;
}

void setDevice(int deviceId) {
    hipSetDevice(deviceId);
}

size_t getFreeMemory() {
    size_t free, total;
    hipMemGetInfo(&free, &total);
    return free;
}

size_t getTotalMemory() {
    size_t free, total;
    hipMemGetInfo(&free, &total);
    return total;
}

void synchronize() {
    hipDeviceSynchronize();
}

void* allocate(size_t size) {
    void* ptr;
    hipMalloc(&ptr, size);
    return ptr;
}

void free(void* ptr) {
    hipFree(ptr);
}

void memcpyHtoD(void* dst, const void* src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void memcpyDtoH(void* dst, const void* src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

void memcpyAsync(void* dst, const void* src, size_t size, hipStream_t stream) {
    hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, stream);
}

}

}
