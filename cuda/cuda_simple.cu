#include "hip/hip_runtime.h"
#include "helix/cuda/cuda_simple.hpp"
#include "helix/common/utils.hpp"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cmath>
#include <cstring>

namespace helix {

// CUDA utility implementations
namespace cuda_simple {

bool isAvailable() {
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    return error == hipSuccess && deviceCount > 0;
}

int getDeviceCount() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    return deviceCount;
}

void setDevice(int deviceId) {
    hipSetDevice(deviceId);
}

size_t getFreeMemory() {
    size_t free, total;
    hipMemGetInfo(&free, &total);
    return free;
}

void synchronize() {
    hipDeviceSynchronize();
}

void* allocate(size_t size) {
    void* ptr;
    hipError_t error = hipMalloc(&ptr, size);
    if (error != hipSuccess) {
        throw HelixException("CUDA memory allocation failed");
    }
    return ptr;
}

void free(void* ptr) {
    if (ptr) {
        hipFree(ptr);
    }
}

void memcpyHtoD(void* dst, const void* src, size_t size) {
    hipError_t error = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        throw HelixException("CUDA memcpy HtoD failed");
    }
}

void memcpyDtoH(void* dst, const void* src, size_t size) {
    hipError_t error = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        throw HelixException("CUDA memcpy DtoH failed");
    }
}

}

// CudaIndexFlatSimple implementation
CudaIndexFlatSimple::CudaIndexFlatSimple(dim_t dimension, MetricType metric)
    : dimension_(dimension), metric_(metric), ntotal_(0), trained_(false),
      d_vectors_(nullptr), d_queries_(nullptr), d_distances_(nullptr), 
      d_indices_(nullptr), gpu_memory_allocated_(0) {
    
    if (!cuda_simple::isAvailable()) {
        throw HelixException("CUDA not available");
    }
    
    allocateGpuMemory();
}

CudaIndexFlatSimple::~CudaIndexFlatSimple() {
    freeGpuMemory();
}

void CudaIndexFlatSimple::allocateGpuMemory() {
    // Allocate GPU memory for vectors (max 1M vectors)
    size_t maxVectors = 1000000;
    size_t vectorSize = maxVectors * dimension_ * sizeof(float);
    size_t querySize = dimension_ * sizeof(float);
    size_t distanceSize = maxVectors * sizeof(float);
    size_t indexSize = maxVectors * sizeof(idx_t);
    
    d_vectors_ = static_cast<float*>(cuda_simple::allocate(vectorSize));
    d_queries_ = static_cast<float*>(cuda_simple::allocate(querySize));
    d_distances_ = static_cast<float*>(cuda_simple::allocate(distanceSize));
    d_indices_ = static_cast<idx_t*>(cuda_simple::allocate(indexSize));
    
    gpu_memory_allocated_ = vectorSize + querySize + distanceSize + indexSize;
    
    // Initialize indices
    std::vector<idx_t> hostIndices(maxVectors);
    for (idx_t i = 0; i < maxVectors; ++i) {
        hostIndices[i] = i;
    }
    cuda_simple::memcpyHtoD(d_indices_, hostIndices.data(), indexSize);
}

void CudaIndexFlatSimple::freeGpuMemory() {
    cuda_simple::free(d_vectors_);
    cuda_simple::free(d_queries_);
    cuda_simple::free(d_distances_);
    cuda_simple::free(d_indices_);
    gpu_memory_allocated_ = 0;
}

void CudaIndexFlatSimple::train(const float* vectors, idx_t numVectors) {
    if (numVectors == 0) return;
    
    // Store vectors
    vectors_.resize(numVectors * dimension_);
    memcpy(vectors_.data(), vectors, numVectors * dimension_ * sizeof(float));
    
    trained_ = true;
}

void CudaIndexFlatSimple::add(const float* vectors, idx_t numVectors) {
    if (!trained_) {
        throw HelixException("Index must be trained before adding vectors");
    }
    
    if (numVectors == 0) return;
    
    // Append to existing vectors
    size_t oldSize = vectors_.size();
    vectors_.resize(oldSize + numVectors * dimension_);
    memcpy(vectors_.data() + oldSize, vectors, numVectors * dimension_ * sizeof(float));
    
    ntotal_ += numVectors;
}

void CudaIndexFlatSimple::syncToGpu() {
    if (ntotal_ > 0) {
        size_t vectorSize = ntotal_ * dimension_ * sizeof(float);
        cuda_simple::memcpyHtoD(d_vectors_, vectors_.data(), vectorSize);
    }
}

SearchResults CudaIndexFlatSimple::search(const float* query, idx_t k) const {
    if (k <= 0 || k > ntotal_) {
        throw HelixException("Invalid k value for search");
    }
    
    if (ntotal_ == 0) {
        return SearchResults();
    }
    
    // Sync data to GPU
    const_cast<CudaIndexFlatSimple*>(this)->syncToGpu();
    
    // Copy query to GPU
    cuda_simple::memcpyHtoD(d_queries_, query, dimension_ * sizeof(float));
    
    // Compute distances on GPU (simplified implementation)
    // In a real implementation, you'd have optimized CUDA kernels
    std::vector<float> distances(ntotal_);
    
    // For now, compute distances on CPU and copy to GPU
    for (idx_t i = 0; i < ntotal_; ++i) {
        float dist = 0.0f;
        for (dim_t d = 0; d < dimension_; ++d) {
            float diff = query[d] - vectors_[i * dimension_ + d];
            dist += diff * diff;
        }
        distances[i] = std::sqrt(dist);
    }
    
    // Copy distances to GPU
    cuda_simple::memcpyHtoD(d_distances_, distances.data(), ntotal_ * sizeof(float));
    
    // Find top-k (simplified implementation)
    std::vector<std::pair<float, idx_t>> distancePairs;
    for (idx_t i = 0; i < ntotal_; ++i) {
        distancePairs.push_back({distances[i], i});
    }
    
    std::sort(distancePairs.begin(), distancePairs.end());
    
    // Create results
    SearchResults result(k);
    
    for (idx_t i = 0; i < k; ++i) {
        result.results.emplace_back(distancePairs[i].second, distancePairs[i].first);
    }
    
    return result;
}

}
